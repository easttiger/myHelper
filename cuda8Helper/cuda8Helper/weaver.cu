

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
const int lena = 83;
const int lenb = 1543;
float a[ lena ], b[ lenb ], x[ lena ], xnew[ lena ], tau[ lenb ];
int De[ lena * lenb ];
int tDe[ lenb * lena ];

inline float sum(float* a, const int lena) {
  float res = *a;
  for(int i = 1; i < lena; ++i) {
    res += a[ i ];
  }
  return res;
}
void transpose(int* tm, int* m, const int nrow, const int ncol) {
  for(int i = 0; i < nrow; ++i) {
    for(int j = 0; j < ncol; ++j) {
      tm[ j * nrow + i ] = m[ i * ncol + j ];
    }
  }
}
float M_v(int* M, int row, float* v, const int len) {
  float res = 0.;
  int* p = M + row * len;
  for(int i = 0; i < len; ++i) {
    if(*p++) res += v[ i ];
  }
  return res;
}


void cpuweaver(float tol) {
  float sa = sum(a, lena);
  for(int i = 0; i < lena; ++i) x[ i ] = a[ i ] / sa;
  float m = sa + sum(b, lenb);

  int iter = 0;
  float e = 999, tmp;
  for(iter = 0; e > 1e-15; ++iter) {
    for(int j = 0; j < lenb; ++j) {
      tau[ j ] = b[ j ] / M_v(tDe, j, x, lena);
    }
    for(int i = 0; i < lena; ++i) {
      xnew[ i ] = a[ i ] / (m - M_v(De, i, tau, lenb));
    }
    e = 0.;
    for(int i = 0; i < lena; ++i) {
      tmp = fabs(xnew[ i ] - x[ i ]);
      if(e < tmp) e = tmp;
    };
    for(int i = 0; i < lena; ++i) {
      x[ i ] = xnew[ i ];
    }
  }
}


void test_weaver() {
  FILE* fp;
  //load a
  fp = fopen("C:/Users/easttiger/Dropbox/Subjects/W/Weaver Paper Reading list/Hunter matlab code/a.txt", "r");
  for(int i = 0; i < lena; ++i) {
    fscanf(fp, "%f\n", a + i);
  }
  fclose(fp);
  //load b
  fp = fopen("C:/Users/easttiger/Dropbox/Subjects/W/Weaver Paper Reading list/Hunter matlab code/b.txt", "r");
  for(int i = 0; i < lenb; ++i) {
    fscanf(fp, "%f\n", b + i);
  }
  fclose(fp);
  //load b
  fp = fopen("C:/Users/easttiger/Dropbox/Subjects/W/Weaver Paper Reading list/Hunter matlab code/tDe.txt", "r");
  for(int j = 0; j < lenb; ++j) {
    for(int i = 0; i < lena - 1; ++i) {
      fscanf(fp, "%d ", tDe + j * lena + i);
    }
    fscanf(fp, "%d\n", tDe + (j + 1) * lena - 1);
  }
  fclose(fp);

  //initialize De as transpose of tDe
  transpose(De, tDe, lenb, lena);


  cpuweaver(0.00000001);
  for(int i = 0; i < lena; ++i) {
    printf("%f,", x[ i ]);
  }
}