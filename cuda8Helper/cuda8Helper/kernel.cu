#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "myCuda.cuh"
#include <stdio.h>

__device__ float logit1(float x) {
  return expf(x) / (1. + expf(x));
}
__global__ void logit(float* y, float* x, int leng) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < leng) {
    y[ i ] = logit1(x[ i ]);
  }
}

int main() {
  const int N = 1024;
  float* x; hipMallocManaged(&x, N * sizeof(float));
  float* y; hipMallocManaged(&y, N * sizeof(float));
  for(int i = 0; i < N; ++i) {
    x[ i ] = (i - N / 2.) / N;
  }
  logit <<<1, N>>>(y, x, N);
  hipDeviceSynchronize();
  for(int i = 0; i < N; ++i) {
    printf("logit(%f)=%f\n", x[ i ], y[ i ]);
  }
  hipFree(x);
  hipFree(y);
  system("pause");
  return  0;
}

