
#include "hip/hip_runtime.h"
#include ""
#include "myCuda.cuh"
#include <stdio.h>
using namespace myCuda::math;

int unified_memory() {
  const int N = 4;
  float* x; hipMallocManaged(&x, N * sizeof(float));
  float* y; hipMallocManaged(&y, N * sizeof(float));
  for(int i = 0; i < N; ++i) {
    x[ i ] = (i - N / 2.) / N;
  }
  logit << <1, N >> >(y, x, N);
  hipDeviceSynchronize();
  for(int i = 0; i < N; ++i) {
    printf("logit(%f)=%f\n", x[ i ], y[ i ]);
  }
  hipFree(x);
  hipFree(y);
  return 0;
}

#define IDX2F(i,j,ld)((((j)-1)*(ld)+((i)-1)))
static __inline__ void modify(hipblasHandle_t handle, float* m, int ldm, int n, int p, int q, float alpha, float beta) {
  hipblasSscal(handle, n-p+1, &alpha, &m[IDX2F(p,q,ldm)], ldm);
  hipblasSscal(handle, ldm-p+1, &beta, &m[IDX2F(p,q,ldm)], 1);
}

int cublas_test() {
  const int M = 6;
  const int N = 5;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float* devPtrA;
  float* a = 0;
  a = (float*)malloc(M * N * sizeof(*a));
  for(j = 1; j <= N; ++j) {
    for(i = 1; i <= M; ++i) {
      a[ IDX2F(i, j, M) ] = (float)((i - 1) * M + j);
    }
  }
  return 0;
}

int main() {
  cout << "unified_memory : " << unified_memory() << endl;
  system("pause");
  return  0;
}
