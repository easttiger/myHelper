#include "hip/hip_runtime.h"
#include ""
#include "myCuda.cuh"
#include <stdio.h>
using namespace myCuda::math;

int unified_memory() {
  const int N = 4;
  float* x; hipMallocManaged(&x, N * sizeof(float));
  float* y; hipMallocManaged(&y, N * sizeof(float));
  for(int i = 0; i < N; ++i) {
    x[ i ] = (i - N / 2.) / N;
  }
  logit <<<1, N >>>(y, x, N);
  hipDeviceSynchronize();
  for(int i = 0; i < N; ++i) {
    printf("logit(%f)=%f\n", x[ i ], y[ i ]);
  }
  hipFree(x);
  hipFree(y);
  return 0;
}

#define IDX2F(i,j,ld)((((j)-1)*(ld)+((i)-1)))
static __inline__ void modify(hipblasHandle_t handle, float* m, int ldm, int n, int p, int q, float alpha, float beta) {
  hipblasSscal(handle, n-p+1, &alpha, &m[IDX2F(p,q,ldm)], ldm);
  hipblasSscal(handle, ldm-p+1, &beta, &m[IDX2F(p,q,ldm)], 1);
}

int cublas_test() {
  const int M = 6;
  const int N = 5;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float* devPtrA;
  float* a = 0;
  a = (float*)malloc(M * N * sizeof(*a));
  for(j = 1; j <= N; ++j) {
    for(i = 1; i <= M; ++i) {
      a[ IDX2F(i, j, M) ] = (float)((i - 1) * M + j);
    }
  }
  return 0;
}

void test_random_number() {
  thrust::device_vector<float> d(10);
  myCuda::ran::runif(d);
  thrust::copy(d.begin(), d.end(), std::ostream_iterator<float>(std::cout, "\n"));
}

void test_weaver();

int main() {
  myCuda::gpuInfo();
  test_random_number();
  test_weaver();
  system("pause");
  return  0;
}
