
#include "hip/hip_runtime.h"
#include ""
#include "myCuda.cuh"
#include <stdio.h>
using namespace myCuda::math;
int main() {
  const int N = 1024;
  float* x; hipMallocManaged(&x, N * sizeof(float));
  float* y; hipMallocManaged(&y, N * sizeof(float));
  for(int i = 0; i < N; ++i) {
    x[ i ] = (i - N / 2.) / N;
  }
  logit <<<1, N>>>(y, x, N);
  hipDeviceSynchronize();
  for(int i = 0; i < N; ++i) {
    printf("logit(%f)=%f\n", x[ i ], y[ i ]);
  }
  hipFree(x);
  hipFree(y);
  system("pause");
  return  0;
}

