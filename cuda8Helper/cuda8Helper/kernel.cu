#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "myCuda.cuh"
#include <stdio.h>
using namespace myCuda::math;
const int lena = 83;
const int lenb = 1543;
float a[ lena ], b[ lenb ], x[ lena ], xnew[ lena ], tau[ lenb ];
int De[ lena * lenb ];
int tDe[ lenb * lena ];

int unified_memory() {
  const int N = 4;
  float* x; hipMallocManaged(&x, N * sizeof(float));
  float* y; hipMallocManaged(&y, N * sizeof(float));
  for(int i = 0; i < N; ++i) {
    x[ i ] = (i - N / 2.) / N;
  }
  logit <<<1, N >>>(y, x, N);
  hipDeviceSynchronize();
  for(int i = 0; i < N; ++i) {
    printf("logit(%f)=%f\n", x[ i ], y[ i ]);
  }
  hipFree(x);
  hipFree(y);
  return 0;
}

#define IDX2F(i,j,ld)((((j)-1)*(ld)+((i)-1)))
static __inline__ void modify(hipblasHandle_t handle, float* m, int ldm, int n, int p, int q, float alpha, float beta) {
  hipblasSscal(handle, n-p+1, &alpha, &m[IDX2F(p,q,ldm)], ldm);
  hipblasSscal(handle, ldm-p+1, &beta, &m[IDX2F(p,q,ldm)], 1);
}

int cublas_test() {
  const int M = 6;
  const int N = 5;
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float* devPtrA;
  float* a = 0;
  a = (float*)malloc(M * N * sizeof(*a));
  for(j = 1; j <= N; ++j) {
    for(i = 1; i <= M; ++i) {
      a[ IDX2F(i, j, M) ] = (float)((i - 1) * M + j);
    }
  }
  return 0;
}

void test_random_number() {
  thrust::device_vector<float> d(10);
  myCuda::ran::runif(d);
  thrust::copy(d.begin(), d.end(), std::ostream_iterator<float>(std::cout, "\n"));
}

inline float sum(float* a, const int lena) {
  float res = *a;
  for(int i = 1; i < lena; ++i) {
    res += a[ i ];
  }
  return res;
}
void transpose(int* tm, int* m, const int nrow, const int ncol) {
  for(int i = 0; i < nrow; ++i) {
    for(int j = 0; j < ncol; ++j) {
      tm[ j * nrow + i ] = m[ i * ncol + j ];
    }
  }
}
float M_v(int* M, int row, float* v, const int len) {
  float res = 0.;
  int* p = M + row * len;
  for(int i = 0; i < len; ++i) {
    if(*p++) res += v[i];
  }
  return res;
}

void cpuweaver(float tol) {
  float sa = sum(a, lena);
  for(int i = 0; i < lena; ++i) x[ i ] = a[ i ] / sa;
  float m = sa + sum(b, lenb);
  
  int iter = 0;
  float e = 999, tmp;
  for(iter = 0; e > 1e-15; ++iter) {
    for(int j = 0; j < lenb; ++j) {
      tau[ j ] = b[ j ] / M_v(tDe, j, x, lena);
    }
    for(int i = 0; i < lena; ++i) {
      xnew[ i ] = a[ i ] / (m - M_v(De, i, tau, lenb));
    }
    e = 0.;
    for(int i = 0; i < lena; ++i) {
      tmp = fabs(xnew[ i ] - x[ i ]);
      if(e < tmp) e = tmp;
    };
    for(int i = 0; i < lena; ++i) {
      x[ i ] = xnew[ i ];
    }
  }
}

void test_weaver() {
  FILE* fp;
  //load a
  fp = fopen("C:/Users/easttiger/Dropbox/Subjects/W/Weaver Paper Reading list/Hunter matlab code/a.txt", "r");
  for(int i = 0; i < lena; ++i) {
    fscanf(fp, "%f\n", a + i);
  }
  fclose(fp);
  //load b
  fp = fopen("C:/Users/easttiger/Dropbox/Subjects/W/Weaver Paper Reading list/Hunter matlab code/b.txt", "r");
  for(int i = 0; i < lenb; ++i) {
    fscanf(fp, "%f\n", b + i);
  }
  fclose(fp);
  //load b
  fp = fopen("C:/Users/easttiger/Dropbox/Subjects/W/Weaver Paper Reading list/Hunter matlab code/tDe.txt", "r");
  for(int j = 0; j < lenb; ++j) {
    for(int i = 0; i < lena - 1; ++i) {
      fscanf(fp, "%d ", tDe + j * lena + i);
    }
    fscanf(fp, "%d\n", tDe + (j + 1) * lena - 1);
  }
  fclose(fp);

  //initialize De as transpose of tDe
  transpose(De, tDe, lenb, lena);

  
  cpuweaver(0.00000001);
  for(int i = 0; i < lena; ++i) {
    printf("%f,", x[ i ]);
  }
}


int main() {
  myCuda::gpuInfo();
  test_random_number();
  test_weaver();
  system("pause");
  return  0;
}
