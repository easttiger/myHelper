#include "hip/hip_runtime.h"
#include "myCuda.cuh"

namespace myCuda {
#define Dim(x, type, length) \
  type * x; hipMallocManaged(&x, sizeof(type)* length)

  void gpuInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for(int i = 0; i < nDevices; ++i) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device %d:\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
      printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
      printf("  Peak Memory Bandwith (GB/s): %f\n", 2. * prop.memoryClockRate * (prop.memoryBusWidth / 8.) / 1.e6);
    }
  }


  namespace print {
    __global__ void print_float(float* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%f,", x[ i ]);
      }
    }

    __global__ void print_int(int* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%d,", x[ i ]);
      }
    }

    __global__ void print_double(double* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%lf,", x[ i ]);
      }
    }

    __global__ void print_long(long* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%ld,", x[ i ]);
      }
    }

    __global__ void print_char(char* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%s,", x[ i ]);
      }
    }

    __global__ void print_cstr(char** x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%s,", x[ i ]);
      }
    }
  }


  namespace math {
    __device__ float logit1(const float x) {
      return expf(x) / (1. + expf(x));
    }

    __global__ void logit(float* y, const float* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        y[ i ] = logit1(x[ i ]);
      }
    }
  }
}
