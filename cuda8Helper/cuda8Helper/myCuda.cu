#include "hip/hip_runtime.h"
#include "myCuda.cuh"

namespace myCuda {
  void gpuInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for(int i = 0; i < nDevices; ++i) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device %d:\n",i);
      printf("  Device name: %s\n", prop.name);
      printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
      printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
      printf("  Peak Memory Bandwith (GB/s): %f\n", 2. * prop.memoryClockRate * (prop.memoryBusWidth / 8.) / 1.e6);      
    }
  }
  namespace math {
    __device__ float logit1(const float x) {
      return expf(x) / (1. + expf(x));
    }

    __global__ void logit(float* y, const float* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        y[ i ] = logit1(x[ i ]);
      }
    }
  }
}
