#include "hip/hip_runtime.h"
#include "myCuda.cuh"

namespace myCuda {
#define Dim(x, type, length) \
  type * x; hipMallocManaged(&x, sizeof(type)* length)

  void gpuInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for(int i = 0; i < nDevices; ++i) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device %d:\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
      printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
      printf("  Peak Memory Bandwith (GB/s): %f\n", 2. * prop.memoryClockRate * (prop.memoryBusWidth / 8.) / 1.e6);
      printf("  Max block size: %d\n", prop.maxThreadsPerBlock);
      printf("  Max num blocks: %d\n", prop.maxGridSize);
      printf("  Warp size: %d\n", prop.warpSize);
    }
  }


  namespace print {
    __global__ void print_float(float* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%f,", x[ i ]);
      }
    }

    __global__ void print_int(int* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%d,", x[ i ]);
      }
    }

    __global__ void print_double(double* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%lf,", x[ i ]);
      }
    }

    __global__ void print_long(long* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%ld,", x[ i ]);
      }
    }

    __global__ void print_char(char* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%s,", x[ i ]);
      }
    }

    __global__ void print_cstr(char** x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        printf("%s,", x[ i ]);
      }
    }
  }


  namespace math {
    __device__ float logit1(const float x) {
      return expf(x) / (1. + expf(x));
    }

    __global__ void logit(float* y, const float* x, int leng) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if(i < leng) {
        y[ i ] = logit1(x[ i ]);
      }
    }
  }

  namespace ran {
    namespace int_hash_fn {
      __host__ __device__
      unsigned int twong7(unsigned int a) { 
        //http://burtleburtle.net/bob/hash/integer.html
        a = (a + 0x7ed55d16) + (a << 12);
        a = (a ^ 0xc761c23c) ^ (a >> 19);
        a = (a + 0x165667b1) + (a << 5);
        a = (a + 0xd3a2646c) ^ (a << 9);
        a = (a + 0xfd7046c5) + (a << 3);
        a = (a ^ 0xb55a4f09) ^ (a >> 16);
        return a;
      }

    }
    namespace gen {      
      struct runif_gen{
        __host__ __device__
        float operator()(unsigned int a) {
          return static_cast<float>(int_hash_fn::twong7(a)) / UINT_MAX;
        }
      };
    }
    __host__ __device__
    void runif(thrust::device_vector<float>& d_v) {      
      thrust::tabulate(d_v.begin(), d_v.end(), gen::runif_gen());
    }
  }
}
